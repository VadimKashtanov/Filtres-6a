#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define pseudo_alea_d_une_grain(i) ((float)((121+(i%1234))*31 % 1001 ) / 1001.0)

//	===============================================================

static __global__ void kerd_nvidia_score_somme(
	uint _t_MODE, uint GRAINE,
	float * y, uint t0, uint T,
	float * score, float * _PRIXS)
{
	uint t = threadIdx.x + blockIdx.x + blockDim.x;
	//
	if (t < T) {
		float s = 0;
		//
		FOR(0, mega_t, MEGA_T) {
			uint depart_plus_t = t_MODE(
				_t_MODE, GRAINE,
				t0, t0+T*MEGA_T,
				t, mega_t,
				T, MEGA_T
			);

			float _y = y[(mega_t*T*1 + 0 + t)*1 + 0];
			float alea = 2*(PSEUDO_ALEA_cuda((t + ((uint)_y % 10001)))%1000)/1000.0-1;
			_y += alea * SCORE_Y_COEF_BRUIT;
			s += cuda_SCORE(
				_y, _PRIXS[depart_plus_t+1], _PRIXS[depart_plus_t],
				alea * SCORE_Y_COEF_BRUIT
			);
		}
		//
		score[t] = s;
		//atomicAdd(&score[0], s);
	}
};

#define HORIZON 32

static __global__ void kerd_addition_horizontale(
	float * vecteur, uint T, float * somme_finale)
{
	uint thx = threadIdx.x;
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	uint __BLOQUE = blockDim.x;
	//
	if (t < T) {
		uint depart_bloque = 2*(t - (t% __BLOQUE));
		//
		for (uint mul=1; mul <= HORIZON;) {
			if (thx % mul == 0) {
				// a = b + c
				uint a = depart_bloque + 2*thx;
				uint b = depart_bloque + 2*thx;
				uint c = depart_bloque + 2*thx + 2*(mul)/2;
				//
				if (!(a < T)) assert(0);
				if (!(b < T)) assert(0);
				if (!(c < T)) assert(0);
				//
				vecteur[a] = vecteur[b] + vecteur[c];
			}
			__syncthreads();
			mul *= 2;
		}
		//
		if (thx == 0) atomicAdd(&somme_finale[0], vecteur[depart_bloque+0]);
	};
};

float nvidia_somme_score(float * y, uint depart, uint T, uint _t_MODE, uint GRAINE)
{
	ASSERT(T % (HORIZON*2) == 0);
	//
	float * somme_score__d = cudalloc<float>(T);
	float * somme_score_finale__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(somme_score_finale__d, 0, sizeof(float)*1));
	CONTROLE_CUDA(hipMemset(somme_score__d, 0, sizeof(float)*T));
	
	//	--- Calcule du Score ---
	kerd_nvidia_score_somme<<<dim3(KERD(T,1)),dim3(1)>>>(
		_t_MODE, GRAINE,
		y, depart, T,
		somme_score__d, cuda_MARCHEE_DE_TRADE
	);
	ATTENDRE_CUDA();

	//	--- Somme Horizontale ---
	kerd_addition_horizontale<<<dim3(KERD(T/2,HORIZON)),dim3(HORIZON)>>>(
		somme_score__d,
		T, somme_score_finale__d
	);
	ATTENDRE_CUDA();

	//	Gpu vers Cpu
	float * somme_score = gpu_vers_cpu<float>(somme_score_finale__d, 1);
	float somme = somme_score[0];
	//
	CONTROLE_CUDA(hipFree(somme_score__d));
	CONTROLE_CUDA(hipFree(somme_score_finale__d));
	free(somme_score);
	//
	return somme;
};

float  nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	return APRES_SCORE(somme / (float)(1 * T * MEGA_T));
};

/*	Regularisation L2
	Pas oublier le Attention mechanisme
*/

//	===============================================================

float d_nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	return dAPRES_SCORE(somme / (float)(1 * T * MEGA_T)) / (float)(1 * T * MEGA_T);
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	uint _t_MODE, uint GRAINE,
	float _dy, float * y, float * dy,
	uint t0, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		FOR(0, mega_t, MEGA_T) {
			uint depart_plus_t = t_MODE(
				_t_MODE, GRAINE,
				t0, t0+T*MEGA_T,
				_t, mega_t,
				T, MEGA_T
			);
			float _y = y[(mega_t*T*1 + 0+_t)*1+0];
			float alea = 2*(PSEUDO_ALEA_cuda((_t + ((uint)_y % 10001)))%1000)/1000.0-1;
			atomicAdd(&dy[(mega_t*T*1 + 0+_t)*1+0], _dy * cuda_dSCORE(
				y[(mega_t*T*1 + 0+_t)*1+0]+alea*SCORE_Y_COEF_BRUIT, _PRIXS[depart_plus_t+1], _PRIXS[depart_plus_t], alea*SCORE_Y_COEF_BRUIT
			));	//atomicAdd car certaines fonction prennent y[-1] comme fin
		}
	}
};

void d_nvidia_somme_score(float d_score, float * y, float * dy, uint depart, uint T, uint _t_MODE, uint GRAINE) {
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		_t_MODE, GRAINE,
		d_score,
		y, dy,
		depart, T,
		cuda_MARCHEE_DE_TRADE
	);
	ATTENDRE_CUDA();
};